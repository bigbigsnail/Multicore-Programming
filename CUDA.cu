
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 4
#define BLOCK_SIZE 2

__global__ 
void matrix_multiplication(int *A, int *B, int *C)
{
	// Block row and column
	int blockX = blockIdx.x;
	int blockY = blockIdx.y;
	// Thread row and column
	int threadX = threadIdx.x;
	int threadY = threadIdx.y;

	int row = blockY * BLOCK_SIZE + threadY;
	int col = blockX * BLOCK_SIZE + threadX;

	int sum = 0;

	__shared__ int tempA[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ int tempB[BLOCK_SIZE][BLOCK_SIZE];

	for (int i = 0; i < (N / BLOCK_SIZE); i++)
	{
		tempA[threadY][threadX] = A[row * N + i * BLOCK_SIZE + threadX];
		tempB[threadY][threadX] = B[(i * BLOCK_SIZE + threadY) * N + col];
		__syncthreads();

		for (int j = 0; j < BLOCK_SIZE; j++)
		{
			sum += tempA[threadY][j] * tempB[j][threadX];
		}
		__syncthreads();

	}

	C[row * N + col] = sum;
}

void print_matrix(int *a)
{
	for(int i = 0; i < N; i++)
	{
		for(int j = 0; j < N; j++)
		{
			printf("%d ", a[i * N + j]);
		}
		printf("\n");
	}
	printf("\n");
}

int main(void)
{
	int *a, *b, *c; //host copy of a, b, c
	int *dev_a, *dev_b, *dev_c; // device copy of a, b, c
	int size = N * N * sizeof( int );
	
	a = ( int* ) malloc(size);
	b = ( int* ) malloc(size);
	c = ( int* ) malloc(size);


	//allocate device copies of a, b
	hipMalloc( ( void** ) &dev_a, size	);
	hipMalloc( ( void** ) &dev_b, size );
	hipMalloc( ( void** ) &dev_c, size );


	// initialize host copies
	for( int i = 0; i < N * N; ++i)
	{
		a[i] = rand( ) % 100;
		b[i] = rand( ) % 100;
	}

	// copy inputs to device
	hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );

	//
	dim3 dimGrid( (N / BLOCK_SIZE),(N / BLOCK_SIZE),1);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE,1);

	// launch add() kernel on GPU, passing parameters
	matrix_multiplication<<< dimGrid, dimBlock>>>( dev_a, dev_b, dev_c);

	hipMemcpy( c, dev_c, size, hipMemcpyDeviceToHost);

	print_matrix(a);
	print_matrix(b);
	print_matrix(c);


	free(a);
	free(b);
	free(c);
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
