
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 512
#define BLOCK_SIZE 32

__global__ 
void matrix_multiplication(int *A, int *B, int *C)
{
	// Block row and column
	int blockX = blockIdx.x;
	int blockY = blockIdx.y;
	// Thread row and column
	int threadX = threadIdx.x;
	int threadY = threadIdx.y;
	
	// The row and column of the result matrix
	int row = blockY * BLOCK_SIZE + threadY;
	int col = blockX * BLOCK_SIZE + threadX;
	
	// local variable to compute result
	int sum = 0;

	__shared__ int tempA[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ int tempB[BLOCK_SIZE][BLOCK_SIZE];
	
	// i stands for i-th block
	for (int i = 0; i < (N / BLOCK_SIZE); i++)
	{
		tempA[threadY][threadX] = A[row * N + i * BLOCK_SIZE + threadX];
		tempB[threadY][threadX] = B[(i * BLOCK_SIZE + threadY) * N + col];
		__syncthreads();

		for (int j = 0; j < BLOCK_SIZE; j++)
		{
			sum += tempA[threadY][j] * tempB[j][threadX];
		}
		__syncthreads();

	}
	
	// Each thread compute an element of result matrix C
	C[row * N + col] = sum;
}

// Print result
void print_matrix(int *a)
{
	for(int i = 0; i < N; i++)
	{
		for(int j = 0; j < N; j++)
		{
			printf("%d ", a[i * N + j]);
		}
		printf("\n");
	}
	printf("\n");
}

int main(void)
{
	int *a, *b, *c; //host copy of a, b, c
	int *dev_a, *dev_b, *dev_c; // device copy of a, b, c
	int size = N * N * sizeof( int );
	
	a = ( int* ) malloc(size);
	b = ( int* ) malloc(size);
	c = ( int* ) malloc(size);


	//allocate device copies of a, b
	hipMalloc( ( void** ) &dev_a, size	);
	hipMalloc( ( void** ) &dev_b, size );
	hipMalloc( ( void** ) &dev_c, size );


	// initialize host copies
	for( int i = 0; i < N * N; ++i)
	{
		a[i] = rand( ) % 100;
		b[i] = rand( ) % 100;
	}

	// copy inputs to device
	hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );

	// block size and number of threads within a block
	dim3 dimGrid( (N / BLOCK_SIZE),(N / BLOCK_SIZE),1);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE,1);

	// launch kernel function on GPU, passing parameters
	matrix_multiplication<<< dimGrid, dimBlock>>>( dev_a, dev_b, dev_c);

	hipMemcpy( c, dev_c, size, hipMemcpyDeviceToHost);

	print_matrix(a);
	print_matrix(b);
	print_matrix(c);


	free(a);
	free(b);
	free(c);
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
